#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

__global__ void mem_trs_test(int * input)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
}

__global__ void mem_trs_test2(int * input, int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < size)
    {
    printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
    }
}

int main()
{
    int size = 150; // array size
    int byte_size = size * sizeof(int); // complete array byte size

    int * h_input;  // host memory pointer
    h_input = (int *) malloc(byte_size); // host memory allocation

    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < size; i++)
    {
        h_input[i] = (int) (rand() & 0xff); // Fill host array with random values
    }

    int * d_input;  // device memory pointer
    hipMalloc((void**)&d_input, byte_size); // device memory allocation

    hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice); // memory cpy from host to device

    dim3 block(32); // 64 threads in x dimesion for each block
    dim3 grid(5);   // 2 blocks in x for whole grid

    mem_trs_test2 << < grid, block >> >(d_input, size); // Now threas are limited in size 
    
    hipDeviceSynchronize();    // wait for device to finish kernel execution 

    // hipMemcpy(h_input, d_input, byte_size, hipMemcpyDeviceToHost); // memory cpy from device to host
    
    // Resource release host & dev
    free(h_input);
    hipFree(d_input);

    hipDeviceReset();

    return 0;
}