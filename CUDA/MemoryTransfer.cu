#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

__global__ void mem_trs_test(int * input)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
}

int main()
{
    int size = 128; // array size
    int byte_size = size * sizeof(int); // complete array byte size

    int * h_input;  // host memory pointer
    h_input = (int *) malloc(byte_size); // host memory allocation

    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < size; i++)
    {
        h_input[i] = (int) (rand() & 0xff); // Fill host array with random values
    }

    int * d_input;  // device memory pointer
    hipMalloc((void**)&d_input, byte_size); // device memory allocation

    hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice); // memory cpy from host to device

    dim3 block(64); // 64 threads in x dimesion for each block
    dim3 grid(2);   // 2 blocks in x for whole grid

    mem_trs_test << < grid, block >> >(d_input);
    
    hipDeviceSynchronize();    // wait for device to finish kernel execution 

    // hipMemcpy(h_input, d_input, byte_size, hipMemcpyDeviceToHost); // memory cpy from device to host
    
    // Resource release host & dev
    free(h_input);
    hipFree(d_input);

    hipDeviceReset();

    return 0;
}