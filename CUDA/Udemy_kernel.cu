#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>


// ============================= [Arrays] ====================================

__global__ void unique_idx_calc_threadIdx(int * input)
{
	int tid = threadIdx.x;
	printf("threadIdx: %d, value: %d \n", tid, input[tid]);
}

// This function allows to calculate unique index for all elements no matter the block it belongs to
__global__ void unique_gid_calculation_2d(int * input)
{
	// For a grid of 4 blocks with 4 threads each
	int tid = threadIdx.x;					// tid	  -> Is the thread id within a block
	
	int block_offset = blockIdx.x * blockDim.x;	// offset -> blockId.x [0~3] * block dim.x = 4

	int row_offset = blockDim.x * gridDim.x * blockIdx.y;
	 
	int gid = tid + row_offset + block_offset; // gid    -> id in block + row_offset + block_offset
	printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, gid: %d, value: %d \n",
		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

int main()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 26, 12, 342, 56, 44, 99 };

	for (int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}
	printf("\n\n");

	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2, 2);

	unique_gid_calculation_2d << < grid, block >> > (d_data);
	// unique_idx_calc_threadIdx << < grid, block >> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();

	return 0;
}

// =========================== [Hello World] =================================

//__global__ void hello_cuda()
//{
//	printf("Hello CUDA World\n");
//}

//__global__ void print_details()
//{
//	printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, blockDim.x: %d, blockDim.y: %d, blockDim.z: %d, gridDim.x: %d, gridDim.y: %d, gridDim.z: %d\n",
//		threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
//}

//int main()
//{
//	int nx, ny, nz;
//	nx = 4;
//	ny = 4;
//	nz = 4;
//
//	dim3 block(2, 2, 2); 
//	dim3 grid(nx / block.x, ny / block.y, nz / block.z);  
//	print_details << < grid, block >> > ();
//	hipDeviceSynchronize();
//
//	hipDeviceReset();
//
//	return 0;
//}
