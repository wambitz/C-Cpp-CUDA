#include "hip/hip_runtime.h"


#include <iostream>

using namespace std;


__global__ void runningSum(int * d)    
{
    int threads = blockDim.x;
    int tid = threadIdx.x;

    // tc - total numer of threads allowed
    for (int tc = threads, step = 1; tc > 0; step *= 2)
    {
        if (tid < tc)
        {
            d[tid+step] += d[tid];
        }
        tc -= step;
    }
}

int main(int argc, char const *argv[])
{
    const int count = 16;
    const int size = count * sizeof(int);

    int * d;
    hipMalloc((void**)&d, size);

    int h[count];
    for (int i = 0; i < count; i++)
    {
        h[i] = i + 1;
    }

    hipMemcpy(d, h, size, hipMemcpyHostToDevice);
    runningSum <<< 1, count-1>>>(d);
    hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < count; i++)
    {
        cout << h[i] << '\t';
    }

    hipFree(d);
    
    return 0;
}