#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"

#define _USE_MATH_DEFINES
#include <iostream>
#include <cmath>

using namespace std;


__device__ __host__ __inline__ float N(float x)
{
    return 0.5 + 0.5 * erf(x * M_SQRT1_2);
}

__device__ __host__ void price((float k, float s, float t, float r, float v, float * c, float * p) 
{
    float srt = v * sqrtf(t);
    float d1 = (logf(s/k)+(r+0.5*v*v)*t) / srt;
    float d2 = d1 - srt;
    *c = N(d1)*s - N(d2)*kert;
    *p = kert - s + *c;
}

__global__ void price(float * k, float * s, float * t, float * r, float * v, float * c, float * p) 
{
    int idx = threadIdx.x
    price(k[idx], s[idx], t[idx], r[idx], v[idx], &c[idx], &p[idx]);
}


int main(int argc, char const *argv[])
{
    const int count = 512;
    float * args[5];

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, CURAND_RNG_PESUDO_MTGP32);

    for (int i = 0; i < 5; i++)
    {
        hipMalloc((void**)&args[i], size);
        hiprandGenerateUniform(gen, args[i], count);
    }

    float * dc, * dp;
    hipMalloc((void**)&dc, size);
    hipMalloc((void**)&dp, size);
    
    price<<<1, count>>>(args[0], args[1], args[2], args[3], args[4], dc, dp);

    hipFree(args);
    hipFree(dc);
    hipFree(dp);
    
    return 0;
}