//////////////////////////////////////////////////////////////////////////
// Date: April 4th, 2019
// File: timing.cu
//////////////////////////////////////////////////////////////////////////

////////////////////////// DEPENDENCIES //////////////////////////////
 
#include "hip/hip_runtime.h"


#include <stdio.h>
// for random initialize
#include <stdlib.h>
#include <time.h>
// for memset
#include <cstring>

void sum_array_cpu(int * a, int * b, int * c, int size)
{
    // printf("CPU values \n");
    for (int i = 0; i < size; i++)
    {
        c[i] = a[i] + b[i];
        // printf("c[%d] -> %d = a[%d] -> %d + b[%d] -> %d \n", i, c[i], i, a[i], i, b[i]);
    }
    //printf("\n");
}

// compare arrays
void compare_arrays(int * a, int * b, int size)
{
    for (int i = 0; i < size; i++)
    {
        if (a[i] != b[i])
        {
            printf("Arrays are different, a[%d] -> %d != b[%d] -> %d \n", i, a[i], i, b[i]);
            return;
        }
    }
    printf("Arrays are  the same \n");
}

////////////////////////// GPU FUNCTIONS //////////////////////////////

__global__ void sum_array_gpu(int * a, int * b, int * c, int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < size)
    {
        c[gid] = a[gid] + b[gid];
        // printf("c[%d] -> %d = a[%d] -> %d + b[%d] -> %d \n", gid, c[gid], gid, a[gid], gid, b[gid]);
    }
}

int main()
{
    // Dimensions and sizes
    const int size = 1 << 25;;                  // number of all threads in grid
    const int block_size = 128;              // size block, each block has threads
    const int NO_BYTES = size * sizeof(int); // grid size un bytes    

    dim3 block(block_size);                  // 128 threads per block
    dim3 grid( (size/block.x) + 1);          // (10000 / 128) + 1  -> 79 blocks of 128 threas each 
                                             // this ensures we hace enough mem for all threads 

    // host pointers & mem alloc;
    int * h_a = (int *) malloc(NO_BYTES);
    int * h_b = (int *) malloc(NO_BYTES);
    int * gpu_result = (int *) malloc(NO_BYTES); // To compare results
    int * cpu_result = (int *) malloc(NO_BYTES);

    // device pointers & mem alloc
    int * d_a;
    int * d_b;
    int * d_c;

    hipMalloc((void**)&d_a, NO_BYTES);
    hipMalloc((void**)&d_b, NO_BYTES);
    hipMalloc((void**)&d_c, NO_BYTES);

    // time variables
    clock_t cpu_start;
    clock_t gpu_start;
    clock_t htod_start;
    clock_t dtoh_start;

    clock_t cpu_end;
    clock_t gpu_end;
    clock_t htod_end;
    clock_t dtoh_end;

    // initialize host pointer
    time_t t;
    srand( (unsigned) time(&t) );
    for (int i = 0; i < size; i++)
    {
        h_a[i] = (int) (rand() & 0xff);
    }

    for (int i = 0; i < size; i++)
    {
        h_b[i] = (int) (rand() & 0xff);
    }

    // initialize result arrays to zero
    memset(gpu_result, 0, NO_BYTES);
    memset(cpu_result, 0, NO_BYTES);
    
    cpu_start = clock();
    sum_array_cpu(h_a, h_b, cpu_result, size);
    cpu_end = clock();

    // Memory transfer from host to device
    htod_start = clock();
    hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);
    htod_end = clock();

    // grid = 79 blocks
    // block = 128 threads per block 
    gpu_start = clock();
    sum_array_gpu <<<grid, block>>> (d_a, d_b, d_c, size);

    // block execution until finish
    hipDeviceSynchronize();
    gpu_end = clock();

    // Memory transfer from device to host
    dtoh_start = clock();
    hipMemcpy(gpu_result, d_c, NO_BYTES, hipMemcpyDeviceToHost);
    dtoh_end = clock();
    
    // array comparison
    compare_arrays(gpu_result, cpu_result, size);

    // Printing time differente / execution time
    printf("Sum array CPU execution time: %4.6f \n", (double) ((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));
    printf("Sum array GPU execution time: %4.6f \n", (double) ((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));
    printf("Host to Device Mem Transfer execution time: %4.6f \n", (double) ((double)(htod_end - htod_start) / CLOCKS_PER_SEC));
    printf("Device to Host Mem Transfer execution time: %4.6f \n", (double) ((double)(dtoh_end - dtoh_start) / CLOCKS_PER_SEC));
    printf("Sum array GPU total execution time: %4.6f \n", (double) ((double)(dtoh_end - htod_start) / CLOCKS_PER_SEC));
    
    // host release mem
    free(h_a);
    free(h_b);
    free(cpu_result);
    free(gpu_result);

    // device release mem
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipDeviceReset();

    return 0;
}

