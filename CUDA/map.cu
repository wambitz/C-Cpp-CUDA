#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"

#include <ctime>
#include <cstdio>
#include <iostream>

using namespace std;


__global__ void addTen(float * d, int count) 
{
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y + threadIdx.z * blockDim.x * blockDim.y;
    int blockPostInGrid = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
    
    // int blockOffset = threadsPerBlock * blockIdx.x
    // int rowOffset = threadsPerBlock * gridDim.x *  blockIdx.y;
    
    int tid = blockPostInGrid * threadsPerBlock + threadPosInBlock;
    
    if (tid < count)
    {
        d[tid] += 10;
    }
    
}

int main(int argc, char const *argv[])
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

    const int count = 123456;
    const int size = count * sizeof(float);

    float * d;
    float h[count];

    hipMalloc((void**)&d, size);
    hiprandGenerateUniform(gen, d, count);

    dim3 block(8, 8, 8);
    dim3 grid(16, 16);

    addTen<<<grid, block>>>(d, count);  

    hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    hipFree(d);
    
    for (int i = 0; i < 100; i++)   
    {
        cout << h[i] << endl;
    }

    return 0;
}

