#include "hip/hip_runtime.h"


#include <iostream>


using namespace std;

__global__ void sumSingleBlock(int * d) 
{
    int tid = threadIdx.x;

    // tc - numer of participating threads;
    for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>=1, stepSize <<= 1)
    {
        //thread must be write
        if (tid < tc)
        {
            int pa = tid * stepSize * 2;
            int pb = pa + stepSize;
            d[pa] += d[pb];
        }
    }
}


int main(int argc, char const *argv[])
{
    const int count = 512;
    const int size = count * sizeof(int);
    int result;

    int h[count];
    for (int i = 0; i < count; i++)
    {
        h[i] = i + 1;
    }

    int * d; 
    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    sumSingleBlock<<<1, count/2>>>(d);

    hipMemcpy(&result, d, size, hipMemcpyDeviceToHost);
    
    cout << "sum is " << result << endl;

    hipFree(d);
    
    return 0;
}