#include "hip/hip_runtime.h"


#include <iostream>

using namespace std;

int main()
{
	int count; 
	hipGetDeviceCount(&count);

	hipDeviceProp_t prop;

	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&prop, i);
		cout << "Device " << i << ": " << prop.name << endl;
		cout << "Compute capability: " << prop.major << "."
			<< prop.minor << endl;
		cout << "Max grid dimensions: (" <<
			prop.maxGridSize[0] << " x " <<
			prop.maxGridSize[1] << " x " <<
			prop.maxGridSize[2] << ")" << endl;

		cout << "Max block dimensions: (" <<
			prop.maxThreadsDim[0] << " x " <<
			prop.maxThreadsDim[1] << " x " <<
			prop.maxThreadsDim[2] << ")" << endl;
	}
	return 0;
}
