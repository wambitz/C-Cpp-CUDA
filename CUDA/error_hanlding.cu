#include "hip/hip_runtime.h"

// #include "common.h"

#include <stdio.h>
// for random initialize
#include <stdlib.h>
#include <time.h>
// for memset
#include <cstring>

__global__ void sum_array_gpu(int * a, int * b, int * c, int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < size)
    {
        c[gid] = a[gid] + b[gid];
        // printf("c[%d] -> %d = a[%d] -> %d + b[%d] -> %d \n", gid, c[gid], gid, a[gid], gid, b[gid]);
    }
}

void check_cuda_error(hipError_t error)
{
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: %s \n", hipGetErrorString(error));
    }
}

void sum_array_cpu(int * a, int * b, int * c, int size)
{
    // printf("CPU values \n");
    for (int i = 0; i < size; i++)
    {
        c[i] = a[i] + b[i];
        // printf("c[%d] -> %d = a[%d] -> %d + b[%d] -> %d \n", i, c[i], i, a[i], i, b[i]);
    }
    //printf("\n");
}

// compare arrays
void compare_arrays(int * a, int * b, int size)
{
    for (int i = 0; i < size; i++)
    {
        if (a[i] != b[i])
        {
            printf("Arrays are different, a[%d] -> %d != b[%d] -> %d \n", i, a[i], i, b[i]);
            return;
        }
    }
    printf("Arrays are  the same \n");
}

int main()
{
    const int size = 1 << 25;                  // number of all threads in grid
    const int block_size = 1024;              // size block, each block has threads
    const int NO_BYTES = size * sizeof(int); // grid size un bytes    



    dim3 block(block_size);                  // 128 threads per block
    dim3 grid( (size/block.x) + 1);      // (10000 / 128) + 1  -> 79 blocks of 128 threas each 
                                             // this ensures we hace enough mem for all threads 

    // host pointers & mem alloc;
    int * h_a = (int *) malloc(NO_BYTES);
    int * h_b = (int *) malloc(NO_BYTES);
    int * h_c = (int *) malloc(NO_BYTES); // To compare results
    int * gpu_result = (int *) malloc(NO_BYTES);

    // device pointers & mem alloc
    int * d_a;
    int * d_b;
    int * d_c;

    check_cuda_error(hipMalloc((void**)&d_a, NO_BYTES));
    hipMalloc((void**)&d_b, NO_BYTES);
    hipMalloc((void**)&d_c, NO_BYTES);

    // initialize host pointer
    time_t t;
    srand( (unsigned) time(&t) );
    for (int i = 0; i < size; i++)
    {
        h_a[i] = (int) (rand() & 0xff);
    }

    for (int i = 0; i < size; i++)
    {
        h_b[i] = (int) (rand() & 0xff);
    }

    memset(gpu_result, 0, NO_BYTES);

    // Memory transfer from host to device
    hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

    // grid = 79 blocks
    // block = 128 threads per block 
    sum_array_gpu <<<grid, block>>> (d_a, d_b, d_c, size);
    sum_array_cpu(h_a, h_b, h_c, size);
    // block execution until finish
    hipDeviceSynchronize();

    // Memory transfer from device to host
    hipMemcpy(gpu_result, d_c, NO_BYTES, hipMemcpyDeviceToHost);
    
    // array comparison
    compare_arrays(gpu_result, h_c, size);
    
    // host release mem
    free(h_a);
    free(h_b);
    free(h_c);
    free(gpu_result);

    // device release mem
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipDeviceReset();

    return 0;
}

