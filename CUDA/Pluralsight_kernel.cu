#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "stdio.h"

__global__ void addArrays(int * a, int * b, int * c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int count = 5;
	const int size = count * sizeof(int);

	int h_a[] = { 1, 2, 3, 4, 5 };
	int h_b[] = { 10, 20, 30, 40, 50 };
	int h_c[count];

	int * d_a;
	int * d_b;
	int * d_c;

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	// hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);

	addArrays << <1, count >> >(d_a, d_b, d_c);

	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
		
	for (int i = 0; i < count; i++)
	{
		printf("%d ", h_c[i]);
	}
	return 0;
}
