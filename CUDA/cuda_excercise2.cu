#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// This function allows to calculate unique index for all elements no matter the block it belongs to
__global__ void unique_gid_calculation_3d(int * data)
{
	// For a grid of 4 blocks with 4 threads each
	int tid = threadIdx.x + blockDim.x * threadIdx.y + threadIdx.z * blockDim.x * blockDim.y;	// tid	  -> Is the thread id within a block

	int num_threads_in_block = blockDim.x * blockDim.y * blockDim.z;
	int block_offset = blockIdx.x * num_threads_in_block;	// offset -> blockId.x [0~3] * block dim.x = 4

	int num_threads_in_row =  num_threads_in_block * gridDim.x;
	int row_offset = num_threads_in_row * blockIdx.y;
	 
    int gid = tid + row_offset + block_offset; // gid    -> id in block + row_offset + block_offset
    
    printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadIdx.x: %d, gid: %d - data: %d \n",
            blockIdx.x, blockIdx.y, blockIdx.z, tid, gid, data[gid]);

}

int main()
{
    // Size
    int size = 64;
    int size_bytes = size * sizeof(int);
    
    // Grid dimensions 
    dim3 block(2, 2, 2);
    dim3 grid(2, 2, 1);

    // Memory allocation
    int * h_input = (int *) malloc(size_bytes);
    int * d_input;

    hipMalloc((void**)&d_input, size_bytes);

    // Rand value assigment
    for (int i = 0; i < size; i++)
    {
        h_input[i] = (int) (rand() & 0xff);
    }

    // Pass host array values to device
    hipMemcpy(d_input, h_input, size_bytes, hipMemcpyHostToDevice);

    // Call kernel
    unique_gid_calculation_3d << <grid, block>> > (d_input);

    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Release memory
    free(h_input);
    hipFree(d_input);

    hipDeviceReset();

    return 0;
}