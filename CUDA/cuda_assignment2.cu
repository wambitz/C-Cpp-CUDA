/////////////////////////////////////////////////////////////////////
// Date: April 4th, 2019
// File: cuda_assignment.cu
/////////////////////////////////////////////////////////////////////

////////////////////////// DEPENDENCIES /////////////////////////////
 
#include "hip/hip_runtime.h"


#include <stdio.h>
// for random initialize
#include <stdlib.h>
#include <time.h>
// for memset
#include <cstring>

////////////////////////////// MACROS ///////////////////////////////

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

////////////////////////// CPU FUNCTIONS ////////////////////////////

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPU Assert: %s %s %d \n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

void sum_array_cpu(int * a, int * b, int * c, int * result, int size)
{
    for (int i = 0; i < size; i++)
    {
        result[i] = a[i] + b[i] + c[i];
    }
}

// compare arrays
void compare_arrays(int * cpu_result, int * gpu_result, int size)
{
    for (int i = 0; i < size; i++)
    {
        if (cpu_result[i] != gpu_result[i])
        {
            printf("Arrays are different, a[%d] -> %d != b[%d] -> %d \n", i, cpu_result[i], i, gpu_result[i]);
            return;
        }
    }
    printf("Arrays are  the same \n");
}

////////////////////////// GPU FUNCTIONS //////////////////////////////

__global__ void sum_array_gpu(int * a, int * b, int * c, int * result, int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < size)
    {
        result[gid] = a[gid] + b[gid] + c[gid];
    }
}

int main()
{
    // Dimensions and sizes
    const int size = 1 << 22;                  // number of all threads in grid
    const int block_size = 512;              // size block, each block has threads
    const int NO_BYTES = size * sizeof(int); // grid size un bytes    

    dim3 block(block_size);                  // 128 threads per block
    dim3 grid( (size/block.x) + 1);          // (10000 / 128) + 1  -> 79 blocks of 128 threas each 
                                             // this ensures we hace enough mem for all threads 

    // host pointers & mem alloc;
    int * h_a = (int *) malloc(NO_BYTES);
    int * h_b = (int *) malloc(NO_BYTES);
    int * h_c = (int *) malloc(NO_BYTES);
    int * gpu_result = (int *) malloc(NO_BYTES); // To compare results
    int * cpu_result = (int *) malloc(NO_BYTES);

    // device pointers & mem alloc
    int * d_a;
    int * d_b;
    int * d_c;
    int * gpu_tmp_result;

    gpuErrChk(hipMalloc((void**)&d_a, NO_BYTES));
    gpuErrChk(hipMalloc((void**)&d_b, NO_BYTES));
    gpuErrChk(hipMalloc((void**)&d_c, NO_BYTES));
    gpuErrChk(hipMalloc((void**)&gpu_tmp_result, NO_BYTES));

    // time variables
    clock_t cpu_start;
    clock_t gpu_start;
    clock_t htod_start;
    clock_t dtoh_start;

    clock_t cpu_end;
    clock_t gpu_end;
    clock_t htod_end;
    clock_t dtoh_end;

    // initialize host pointer
    time_t t;
    srand( (unsigned) time(&t) );
    for (int i = 0; i < size; i++)
    {
        h_a[i] = (int) (rand() & 0xff);
        h_b[i] = (int) (rand() & 0xff);
        h_c[i] = (int) (rand() & 0xff);

        // printf("h_a[%d]: %d, h_b[%d]: %d, h_c[%d]: %d \n", i, h_a[i], i, h_b[i], i, h_c[i]);
    }

    // initialize result arrays to zero
    memset(gpu_result, 0, NO_BYTES);
    memset(cpu_result, 0, NO_BYTES);
    
    cpu_start = clock();
    sum_array_cpu(h_a, h_b, h_c, cpu_result, size);
    cpu_end = clock();

    // Memory transfer from host to device
    htod_start = clock();
    gpuErrChk(hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpy(d_c, h_c, NO_BYTES, hipMemcpyHostToDevice));
    htod_end = clock();

    // grid = 79 blocks
    // block = 128 threads per block 
    gpu_start = clock();
    sum_array_gpu <<<grid, block>>> (d_a, d_b, d_c, gpu_tmp_result, size);

    // block execution until finish
    gpuErrChk(hipDeviceSynchronize());
    gpu_end = clock();

    // Memory transfer from device to host
    dtoh_start = clock();
    gpuErrChk(hipMemcpy(gpu_result, gpu_tmp_result, NO_BYTES, hipMemcpyDeviceToHost));
    dtoh_end = clock();
    
    // array comparison
    compare_arrays(cpu_result, gpu_result, size);

    // Printing time differente / execution time
    printf("Sum array CPU execution time: %4.6f \n", (double) ((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));
    printf("Sum array GPU execution time: %4.6f \n", (double) ((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));
    printf("Host to Device Mem Transfer execution time: %4.6f \n", (double) ((double)(htod_end - htod_start) / CLOCKS_PER_SEC));
    printf("Device to Host Mem Transfer execution time: %4.6f \n", (double) ((double)(dtoh_end - dtoh_start) / CLOCKS_PER_SEC));
    printf("Sum array GPU total execution time: %4.6f \n", (double) ((double)(dtoh_end - htod_start) / CLOCKS_PER_SEC));
    
    // host release mem
    free(h_a);
    free(h_b);
    free(h_c);
    free(cpu_result);
    free(gpu_result);

    // device release mem
    gpuErrChk(hipFree(d_a));
    gpuErrChk(hipFree(d_b));
    gpuErrChk(hipFree(d_c));
    gpuErrChk(hipFree(gpu_tmp_result));

    gpuErrChk(hipDeviceReset());

    return 0;
}

